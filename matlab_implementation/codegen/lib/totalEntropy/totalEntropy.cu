#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
// File: totalEntropy.cu
//
// GPU Coder version                    : 1.3
// CUDA/C/C++ source code generated on  : 08-May-2019 12:21:38
//

// Include Files
#include <cmath>
#include "totalEntropy.h"

// Function Definitions

//
// Calculation of expectation over all trajectorties
// Arguments    : const emxArray_real_T *Sk
//                const emxArray_real_T *del_uk
//                double lambda
// Return Type  : double
//
double totalEntropy(const emxArray_real_T *Sk, const emxArray_real_T *del_uk,
                    double lambda)
{
  double sum1;
  double sum2;
  int i0;
  int i;

  //  Normalization of cost function
  //      Sk = Sk./sum(Sk);\
  sum1 = 0.0;
  sum2 = 0.0;
  i0 = Sk->size[0];
  for (i = 0; i < i0; i++) {
    sum1 += std::exp(-(1.0 / lambda) * Sk->data[i]) * del_uk->data[i];
    sum2 += std::exp(-(1.0 / lambda) * Sk->data[i]);
  }

  return sum1 / sum2;
}

//
// File trailer for totalEntropy.cu
//
// [EOF]
//
