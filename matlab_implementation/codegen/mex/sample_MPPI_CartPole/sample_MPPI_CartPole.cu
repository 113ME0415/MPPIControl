#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * sample_MPPI_CartPole.cu
 *
 * Code generation for function 'sample_MPPI_CartPole'
 *
 */

/* Include files */
#include <stdlib.h>
#include "MWCudaDimUtility.h"
#include <math.h>
#include "rt_nonfinite.h"
#include "sample_MPPI_CartPole.h"
#include "sample_MPPI_CartPole_data.h"

/* Function Declarations */
static void checkCudaError(hipError_t errCode, const char * file, uint32_T line);
static __global__ void sample_MPPI_CartPole_kernel1(real_T dt, real_T dX[4],
  real_T x[400], int32_T i, real_T b_x[4]);
static __global__ void sample_MPPI_CartPole_kernel2(int32_T i, int32_T *b_i);
static __global__ void sample_MPPI_CartPole_kernel3(real_T x[4], int32_T *i,
  real_T b_x[400]);

/* Function Definitions */
static void checkCudaError(hipError_t errCode, const char * file, uint32_T line)
{
  emlrtRTEInfo rtInfo;
  char_T *pn;
  char_T *fn;
  char_T *brk;
  uint32_T len;
  if (errCode != hipSuccess) {
    len = strlen(file);
    pn = (char_T *)calloc(len + 1U, 1U);
    fn = (char_T *)calloc(len + 1U, 1U);
    memcpy(pn, file, len);
    memcpy(fn, file, len);
    brk = strrchr(fn, '.');
    *brk = '\x00';
    brk = strrchr(fn, '/');
    if (brk == NULL) {
      brk = strrchr(fn, '\\');
    }

    if (brk == NULL) {
      brk = fn;
    } else {
      brk++;
    }

    rtInfo.lineNo = static_cast<int32_T>(line);
    rtInfo.colNo = 0;
    rtInfo.fName = brk;
    rtInfo.pName = pn;
    emlrtCUDAError(errCode, hipGetErrorName(errCode), hipGetErrorString
                   (errCode), &rtInfo, emlrtRootTLSGlobal);
  }
}

static __global__ __launch_bounds__(32, 1) void sample_MPPI_CartPole_kernel1
  (real_T dt, real_T dX[4], real_T x[400], int32_T i, real_T b_x[4])
{
  uint32_T threadId;
  int32_T i1;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i1 = static_cast<int32_T>(threadId);
  if (i1 < 4) {
    b_x[i1] = x[i1 + (i << 2)] + dX[i1] * dt;
  }
}

static __global__ __launch_bounds__(32, 1) void sample_MPPI_CartPole_kernel2
  (int32_T i, int32_T *b_i)
{
  uint32_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *b_i = i + 1;
  }
}

static __global__ __launch_bounds__(32, 1) void sample_MPPI_CartPole_kernel3
  (real_T x[4], int32_T *i, real_T b_x[400])
{
  uint32_T threadId;
  int32_T i1;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  i1 = static_cast<int32_T>(threadId);
  if (i1 < 4) {
    b_x[i1 + (*i << 2)] = x[i1];
  }
}

void sample_MPPI_CartPole(real_T k, real_T N, real_T variance, real_T Stk[1000],
  real_T x[400], real_T delta_u[100000], const real_T u[100], real_T R, real_T
  mc, real_T mp, real_T l, real_T g, real_T kd, real_T dt)
{
  int32_T i0;
  int32_T i;
  real_T r;
  real_T dX[4];
  real_T b_u;
  real_T b_x;
  real_T (*gpu_dX)[4];
  real_T (*gpu_x)[400];
  real_T (*b_gpu_x)[4];
  int32_T *gpu_i;
  boolean_T x_dirtyOnCpu;

#define CUDACHECK(errCall)             checkCudaError(errCall, __FILE__, __LINE__)

  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  CUDACHECK(hipMalloc(&gpu_i, 4ULL));
  CUDACHECK(hipMalloc(&b_gpu_x, 32ULL));
  CUDACHECK(hipMalloc(&gpu_x, 3200ULL));
  CUDACHECK(hipMalloc(&gpu_dX, 32ULL));
  x_dirtyOnCpu = true;
  i0 = static_cast<int32_T>((N - 1.0));
  for (i = 0; i < i0; i++) {
    emlrtRandn(&r, 1);
    delta_u[i + 100 * (static_cast<int32_T>(k) - 1)] = variance * r;
    r = u[i];
    b_u = u[i] + delta_u[i + 100 * (static_cast<int32_T>(k) - 1)];

    /*  States of the Pendulum dX(1) is X_dot dX(2) is X double dor, dX(3) is */
    /*  Theta_dot and dX(4) is theta double dot */
    /*  u is the force horizontally applied */
    /*  down position theta=0, up position theta=PI */
    dX[0] = x[1 + (i << 2)];
    b_x = sin(x[2 + (i << 2)]);
    dX[1] = ((b_u - kd * x[1 + (i << 2)]) + mp * sin(x[2 + (i << 2)]) * (l * (x
               [3 + (i << 2)] * x[3 + (i << 2)]) + g * cos(x[2 + (i << 2)]))) /
      (mc + mp * (b_x * b_x));
    dX[2] = x[3 + (i << 2)];
    b_x = sin(x[2 + (i << 2)]);
    dX[3] = ((-(b_u - kd * x[1 + (i << 2)]) * cos(x[2 + (i << 2)]) - mp * l *
              (x[3 + (i << 2)] * x[3 + (i << 2)]) * cos(x[2 + (i << 2)]) * sin
              (x[2 + (i << 2)])) - (mc + mp) * g * sin(x[2 + (i << 2)])) / (l *
      (mc + mp * (b_x * b_x)));
    CUDACHECK(hipMemcpy(gpu_dX, &dX[0], 32ULL, hipMemcpyHostToDevice));
    if (x_dirtyOnCpu) {
      CUDACHECK(hipMemcpy(gpu_x, (void *)x, 3200ULL, hipMemcpyHostToDevice));
      x_dirtyOnCpu = false;
    }

    sample_MPPI_CartPole_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(dt,
      *gpu_dX, *gpu_x, i, *b_gpu_x);
    CUDACHECK(hipGetLastError());
    sample_MPPI_CartPole_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(i,
      gpu_i);
    CUDACHECK(hipGetLastError());
    sample_MPPI_CartPole_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
      (*b_gpu_x, gpu_i, *gpu_x);
    CUDACHECK(hipGetLastError());
    b_u = u[i] + delta_u[i + 100 * (static_cast<int32_T>(k) - 1)];

    /*  p is position of Cart Pole Theta is angle theta_dot is change in */
    /*  angle and p_dot is change in position */
    CUDACHECK(hipMemcpy((void *)x, gpu_x, 3200ULL, hipMemcpyDeviceToHost));
    b_x = cos(x[2 + ((1 + i) << 2)]);
    Stk[static_cast<int32_T>(k) - 1] += ((((6.0 * (x[(1 + i) << 2] * x[(1 + i) <<
      2]) + 12.0 * ((1.0 + b_x) * (1.0 + b_x))) + R * (b_u * b_u)) + 0.1 * (x[3
      + ((1 + i) << 2)] * x[3 + ((1 + i) << 2)])) + 0.1 * (x[1 + ((1 + i) << 2)]
      * x[1 + ((1 + i) << 2)])) * dt;
  }

  /*  end function */
  CUDACHECK(hipFree(*gpu_dX));
  CUDACHECK(hipFree(*gpu_x));
  CUDACHECK(hipFree(*b_gpu_x));
  CUDACHECK(hipFree(gpu_i));

#undef CUDACHECK

}

/* End of code generation (sample_MPPI_CartPole.cu) */
