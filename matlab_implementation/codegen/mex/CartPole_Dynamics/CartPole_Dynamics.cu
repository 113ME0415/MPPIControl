#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * CartPole_Dynamics.cu
 *
 * Code generation for function 'CartPole_Dynamics'
 *
 */

/* Include files */
#include <math.h>
#include "rt_nonfinite.h"
#include "CartPole_Dynamics.h"

/* Function Definitions */
void CartPole_Dynamics(real_T, real_T x_dot, real_T theta, real_T theta_dot,
  real_T u, real_T mc, real_T mp, real_T l, real_T g, real_T kd, real_T dX[4])
{
  real_T b_x;

  /*  States of the Pendulum dX(1) is X_dot dX(2) is X double dor, dX(3) is */
  /*  Theta_dot and dX(4) is theta double dot */
  /*  u is the force horizontally applied */
  /*  down position theta=0, up position theta=PI */
  dX[0] = x_dot;
  b_x = sin(theta);
  dX[1] = ((u - kd * x_dot) + mp * sin(theta) * (l * (theta_dot * theta_dot) + g
            * cos(theta))) / (mc + mp * (b_x * b_x));
  dX[2] = theta_dot;
  b_x = sin(theta);
  dX[3] = ((-(u - kd * x_dot) * cos(theta) - mp * l * (theta_dot * theta_dot) *
            cos(theta) * sin(theta)) - (mc + mp) * g * sin(theta)) / (l * (mc +
    mp * (b_x * b_x)));
}

/* End of code generation (CartPole_Dynamics.cu) */
