#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * totalEntropy.cu
 *
 * Code generation for function 'totalEntropy'
 *
 */

/* Include files */
#include <math.h>
#include "rt_nonfinite.h"
#include "totalEntropy.h"

/* Function Definitions */
real_T totalEntropy(const emxArray_real_T *Sk, const emxArray_real_T *del_uk,
                    real_T lambda)
{
  real_T sum1;
  real_T sum2;
  int32_T i0;
  int32_T i;

  /*  Calculation of expectation over all trajectorties  */
  /*  Normalization of cost function  */
  /*      Sk = Sk./sum(Sk);\ */
  sum1 = 0.0;
  sum2 = 0.0;
  i0 = Sk->size[0];
  for (i = 0; i < i0; i++) {
    sum1 += exp(-(1.0 / lambda) * Sk->data[i]) * del_uk->data[i];
    sum2 += exp(-(1.0 / lambda) * Sk->data[i]);
  }

  return sum1 / sum2;
}

/* End of code generation (totalEntropy.cu) */
