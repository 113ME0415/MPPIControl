#include "hip/hip_runtime.h"
/*
 * Academic License - for use in teaching, academic research, and meeting
 * course requirements at degree granting institutions only.  Not for
 * government, commercial, or other organizational use.
 *
 * cost_function_cartpole.cu
 *
 * Code generation for function 'cost_function_cartpole'
 *
 */

/* Include files */
#include <math.h>
#include "rt_nonfinite.h"
#include "cost_function_cartpole.h"

/* Function Definitions */
real_T cost_function_cartpole(real_T p, real_T p_dot, real_T theta, real_T
  theta_dot, real_T u, real_T dt, real_T R)
{
  real_T a;

  /*  p is position of Cart Pole Theta is angle theta_dot is change in */
  /*  angle and p_dot is change in position */
  a = 1.0 + cos(theta);
  return ((((6.0 * (p * p) + 12.0 * (a * a)) + R * (u * u)) + 0.1 * (theta_dot *
            theta_dot)) + 0.1 * (p_dot * p_dot)) * dt;
}

/* End of code generation (cost_function_cartpole.cu) */
